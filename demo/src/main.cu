#include <iostream>
#include <chrono>
#include <igl/writeOBJ.h>
#include <igl/read_triangle_mesh.h>
#include <igl/vertex_triangle_adjacency.h>

#include "flo/load_mesh.hpp"
#include "flo/host/flo_matrix_operation.hpp"
#include "flo/host/willmore_flow.hpp"
#include "flo/device/vertex_triangle_adjacency.cuh"
#include "flo/host/vertex_triangle_adjacency.hpp"

using namespace Eigen;

template <typename T>
void forward_euler(gsl::span<T> i_x,
                   const gsl::span<const T> i_dx,
                   const double i_t)
{
  std::transform(i_x.begin(), i_x.end(), i_dx.begin(), i_x.begin(),
                 [i_t](T x, T dx)
                 {
                   return x + dx * i_t;
                 });
}

int main()
{
  auto surf = flo::load_mesh("foo.obj");
  auto V = flo::host::array_to_matrix(gsl::make_span(surf.vertices));
  auto F = flo::host::array_to_matrix(gsl::make_span(surf.faces));

  //std::cout<<"V:\n"<<V<<'\n';
  //std::cout<<"F:\n"<<F<<'\n';

  //const auto integrator = [tao=0.95](auto x, const auto dx){
  //  return forward_euler(x, dx, tao);
  //};

  //for (int iter = 0; iter < 3; ++iter)
  //{
  //  std::cout<<"Iteration: "<<iter<<'\n';
  //  surf.vertices = flo::host::willmore_flow(surf.vertices, surf.faces, integrator);
  //}


  
  thrust::device_vector<int> d_face_verts(surf.n_faces() * 3);
  thrust::copy_n((&surf.faces[0][0]), surf.n_faces() * 3, d_face_verts.data());
  thrust::device_vector<int> d_adjacency(surf.n_faces() * 3);
  thrust::device_vector<int> d_valence(surf.n_vertices());
  thrust::device_vector<int> d_cumulative_valence(surf.n_vertices() + 1);

  thrust::tabulate(d_adjacency.begin(), d_adjacency.end(), 
      [] __device__ (int idx) { return idx / 3; });
  auto ptr_tuple = thrust::make_tuple(d_face_verts.data(), d_adjacency.data());
  auto zip_begin = thrust::make_zip_iterator(ptr_tuple);

  auto time_begin = std::chrono::high_resolution_clock::now();
  // The sort is based on the vertex indices
  //thrust::sort_by_key(d_face_verts.begin(), d_face_verts.end(), d_adjacency.begin());
  flo::device::vertex_triangle_adjacency(
      d_face_verts.data(), 
      surf.n_faces(), 
      surf.n_vertices(), 
      d_adjacency.data(), 
      d_valence.data(), 
      d_cumulative_valence.data());
  hipDeviceSynchronize();
  auto time_end = std::chrono::high_resolution_clock::now();

  using namespace std::chrono;
  std::cout<<"Time taken: "<<duration_cast<nanoseconds>(time_end-time_begin).count()<<'\n';
  
  //std::vector<int> h_adjacency(surf.n_faces() * 3);
  //thrust::copy(d_adjacency.begin(), d_adjacency.end(), h_adjacency.begin());
  //auto VF = flo::host::array_to_matrix(gsl::make_span(h_adjacency));

  //std::vector<int> h_cumulative_valence(surf.n_vertices() + 1);
  //thrust::copy(d_cumulative_valence.begin(), d_cumulative_valence.end(), h_cumulative_valence.begin());
  //auto NI = flo::host::array_to_matrix(gsl::make_span(h_cumulative_valence));

  //std::cout<<"VF:\n"<<VF<<'\n';
  //std::cout<<"NI:\n"<<NI<<'\n';


  igl::writeOBJ("bar.obj", V, F);

  return 0;
}
