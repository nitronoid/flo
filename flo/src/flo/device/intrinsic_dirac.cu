#include "hip/hip_runtime.h"
#include "flo/device/intrinsic_dirac.cuh"
#include "flo/device/thread_util.cuh"
#include "flo/device/matrix_operation.cuh"
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/discard_iterator.h>

FLO_DEVICE_NAMESPACE_BEGIN

namespace
{
template <typename T>
__device__ __forceinline__ constexpr T sqr(T&& i_value) noexcept
{
  return i_value * i_value;
}
__device__ real4 hammilton_product(const real3& i_rhs, const real3& i_lhs)
{
  const real a1 = 0.f;
  const real b1 = i_rhs.x;
  const real c1 = i_rhs.y;
  const real d1 = i_rhs.z;
  const real a2 = 0.f;
  const real b2 = i_lhs.x;
  const real c2 = i_lhs.y;
  const real d2 = i_lhs.z;
  // W is last in a vector
  return make_float4(a1 * b2 + b1 * a2 + c1 * d2 - d1 * c2,
                     a1 * c2 - b1 * d2 + c1 * a2 + d1 * b2,
                     a1 * d2 + b1 * c2 - c1 * b2 + d1 * a2,
                     a1 * a2 - b1 * b2 - c1 * c2 - d1 * d2);
}
__device__ real4 hammilton_product(
  real lhs_x, real lhs_y, real lhs_z, real rhs_x, real rhs_y, real rhs_z)
{
  const real a1 = 0.f;
  const real b1 = lhs_x;
  const real c1 = lhs_y;
  const real d1 = lhs_z;
  const real a2 = 0.f;
  const real b2 = rhs_x;
  const real c2 = rhs_y;
  const real d2 = rhs_z;
  // W is last in a vector
  return make_float4(a1 * b2 + b1 * a2 + c1 * d2 - d1 * c2,
                     a1 * c2 - b1 * d2 + c1 * a2 + d1 * b2,
                     a1 * d2 + b1 * c2 - c1 * b2 + d1 * a2,
                     a1 * a2 - b1 * b2 - c1 * c2 - d1 * d2);
}

template <typename T>
__device__ constexpr T reciprocal(T&& i_value) noexcept
{
  return T{1} / i_value;
}

__device__ real4 make_quat(real4 i_value) noexcept
{
  return i_value;
}

__device__ real4 make_quat(real i_value) noexcept
{
  real4 q;
  q.x = 0.f;
  q.y = 0.f;
  q.z = 0.f;
  q.w = i_value;
  return q;
}

template <
  typename T,
  typename = typename std::enable_if<std::is_same<T, real>::value ||
                                     std::is_same<T, real4>::value>::type>
__global__ void
d_to_real_quaternion_matrix(const int* __restrict__ di_rows,
                            const int* __restrict__ di_columns,
                            const T* __restrict__ di_values,
                            const int* __restrict__ di_cumulative_column_size,
                            const int i_nvalues,
                            int* __restrict__ do_rows,
                            int* __restrict__ do_columns,
                            real* __restrict__ do_values)
{
  // Declare one shared memory block
  extern __shared__ uint8_t shared_memory[];
  // Offset our shared memory pointer by the number of values * sizeof(real4)
  int32_t* __restrict__ row_index = (int32_t*)(shared_memory);
  // Offset our shared memory pointer by the number of values * sizeof(int)
  int32_t* __restrict__ col_index = (int32_t*)(row_index + blockDim.x * 4);
  // Create pointers into the block dividing it for the different uses
  real4* __restrict__ quaternion_entry = (real4*)(col_index + blockDim.x * 4);

  // Calculate which entry this thread is transforming
  const int global_id = blockIdx.x * blockDim.x + threadIdx.x;

  // Guard against out of range threads
  if (global_id >= i_nvalues)
    return;

  // Get our block local id for shared memory access
  const int16_t local_id = threadIdx.x * 4 + threadIdx.y;

  if (!threadIdx.y)
  {
    // Read the quaternion entry once
    const real4 quat = make_quat(di_values[global_id]);
    // Copy the quaternion across shared memory so all threads have access
    quaternion_entry[threadIdx.x * 4 + 0] = quat;
    quaternion_entry[threadIdx.x * 4 + 1] = quat;
    quaternion_entry[threadIdx.x * 4 + 2] = quat;
    quaternion_entry[threadIdx.x * 4 + 3] = quat;
    // Read the row index once
    const int row = di_rows[global_id];
    // Copy across shared memory so all threads have access
    row_index[threadIdx.x * 4 + 0] = row;
    row_index[threadIdx.x * 4 + 1] = row;
    row_index[threadIdx.x * 4 + 2] = row;
    row_index[threadIdx.x * 4 + 3] = row;
    // Read the column index once
    const int col = di_columns[global_id];
    // Copy across shared memory so all threads have access
    col_index[threadIdx.x * 4 + 0] = col;
    col_index[threadIdx.x * 4 + 1] = col;
    col_index[threadIdx.x * 4 + 2] = col;
    col_index[threadIdx.x * 4 + 3] = col;
  }
  __syncthreads();
  const uint8_t sign = (0x284E >> (threadIdx.y * 4u)) & 15u;
  const uchar4 loop = quat_loop(threadIdx.y);
  real4 quat;
  quat.x =
    nth_element(quaternion_entry[local_id], loop.x) * sign_from_bit(sign, 0u);
  quat.y =
    nth_element(quaternion_entry[local_id], loop.y) * sign_from_bit(sign, 1u);
  quat.z =
    nth_element(quaternion_entry[local_id], loop.z) * sign_from_bit(sign, 2u);
  quat.w =
    nth_element(quaternion_entry[local_id], loop.w) * sign_from_bit(sign, 3u);

  // Calculate where we're writing to
  const int32_t prev_col_offset =
    di_cumulative_column_size[row_index[local_id] - 1];
  const int32_t curr_col_offset =
    di_cumulative_column_size[row_index[local_id]] - prev_col_offset;
  const int32_t offset = prev_col_offset * 16 +
                         curr_col_offset * 4 * threadIdx.y +
                         (global_id - prev_col_offset) * 4;

  // Use a vector cast to write using a 16 byte instruction
  *reinterpret_cast<real4*>(do_values + offset) = quat;
  int32_t C = col_index[local_id] * 4;
  *reinterpret_cast<int4*>(do_columns + offset) =
    make_int4(C, C + 1, C + 2, C + 3);
  *reinterpret_cast<int4*>(do_rows + offset) =
    make_int4(row_index[local_id] * 4 + threadIdx.y);
}

enum MASK { FULL_MASK = 0xffffffff };
// block dim should be 3*#F, where #F is some number of faces,
// we have three edges per triangle face, and write two values per edge
__global__ void d_intrinsic_dirac_atomic(const real* __restrict__ di_vertices,
                                         const int* __restrict__ di_faces,
                                         const real* __restrict__ di_rho,
                                         const int* __restrict__ di_entry,
                                         const int i_nverts,
                                         const int i_nfaces,
                                         real4* __restrict__ do_values)
{
  // Calculate our global thread index
  const int32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  // The face index is the thread index / 4
  const int32_t fid = tid >> 2;
  // Calculate which lane our thread is in: [0,1,2,3]
  const int8_t lane = tid - fid * 4;

  // Guard against threads that would read/write out of bounds
  if (fid >= i_nfaces)
    return;

  // Load vertex points into registers
  real edge_x, edge_y, edge_z, rho;
  // First three threads read from global memory
  if (lane < 3)
  {
    const int32_t pid = di_faces[i_nfaces * lane + fid];
    edge_x = di_vertices[0 * i_nverts + pid];
    edge_y = di_vertices[1 * i_nverts + pid];
    edge_z = di_vertices[2 * i_nverts + pid];
    rho = di_rho[pid];
  }
  // Convert our 0,1,2 reads into the 1,2,0,1 layout over four threads
  {
    const int8_t source_lane = (lane + 1) - 3 * (lane > 1);
    edge_x = __shfl_sync(FULL_MASK, edge_x, source_lane, 4);
    edge_y = __shfl_sync(FULL_MASK, edge_y, source_lane, 4);
    edge_z = __shfl_sync(FULL_MASK, edge_z, source_lane, 4);
  }
  {
    const int8_t source_lane = lane - 3 * (lane == 3);
    rho = __shfl_sync(FULL_MASK, rho, source_lane, 4);
  }
  // Compute edge vectors from neighbor threads
  // 1-2, 2-0, 0-1, 1-2
  // 2-1, 0-2, 1-0, 2-1
  {
    const int8_t source_lane = (lane + 1) - 3 * (lane == 3);
    edge_x = __shfl_sync(FULL_MASK, edge_x, source_lane, 4) - edge_x;
    edge_y = __shfl_sync(FULL_MASK, edge_y, source_lane, 4) - edge_y;
    edge_z = __shfl_sync(FULL_MASK, edge_z, source_lane, 4) - edge_z;
  }

  // Get the components of the neighboring edge
  const real b_x = __shfl_down_sync(FULL_MASK, edge_x, 1, 4);
  const real b_y = __shfl_down_sync(FULL_MASK, edge_y, 1, 4);
  const real b_z = __shfl_down_sync(FULL_MASK, edge_z, 1, 4);
  const real b_rho = __shfl_down_sync(FULL_MASK, rho, 1, 4);

  // Compute the inverse area (1/-4A == 1/(-4*0.5*x^1/2) == -0.5 * 1/(x^1/2))
  const real inv_area = -0.5f * __frsqrt_rn(sqr(edge_y * b_z - edge_z * b_y) +
                                            sqr(edge_z * b_x - edge_x * b_z) +
                                            sqr(edge_x * b_y - edge_y * b_x));

  const real c = ((1.f / inv_area) * -0.25f) * reciprocal(9.f) * rho * b_rho;
  const real4 img = make_float4(reciprocal(6.f) * (rho * b_x - b_rho * edge_x),
                                reciprocal(6.f) * (rho * b_y - b_rho * edge_y),
                                reciprocal(6.f) * (rho * b_z - b_rho * edge_z),
                                0.f);

  // Compute lower result TODO: Vectorized atomics
  if (lane < 3)
  {
    real4 result =
      hammilton_product(edge_x, edge_y, edge_z, b_x, b_y, b_z) * inv_area + img;
    result.w += c;

    const int32_t address = di_entry[i_nfaces * lane + fid];

    auto out = reinterpret_cast<real*>(do_values + address);
    atomicAdd(out + 0, result.x);
    atomicAdd(out + 1, result.y);
    atomicAdd(out + 2, result.z);
    atomicAdd(out + 3, result.w);
  }

  // Compute upper result
  if (lane < 3)
  {
    real4 result =
      hammilton_product(b_x, b_y, b_z, edge_x, edge_y, edge_z) * inv_area - img;
    result.w += c;

    const int32_t address = di_entry[i_nfaces * (lane + 3) + fid];

    auto out = reinterpret_cast<real*>(do_values + address);
    atomicAdd(out + 0, result.x);
    atomicAdd(out + 1, result.y);
    atomicAdd(out + 2, result.z);
    atomicAdd(out + 3, result.w);
  }
}

struct dirac_diagonal
  : public thrust::unary_function<thrust::tuple<int, const int>, flo::real4>
{
  dirac_diagonal(thrust::device_ptr<const real> di_vertices,
                 thrust::device_ptr<const int> di_faces,
                 thrust::device_ptr<const real> di_face_area,
                 thrust::device_ptr<const real> di_rho,
                 int32_t i_nverts,
                 int32_t i_nfaces)
    : di_vertices(std::move(di_vertices.get()))
    , di_faces(std::move(di_faces.get()))
    , di_face_area(std::move(di_face_area.get()))
    , di_rho(std::move(di_rho.get()))
    , nverts(std::move(i_nverts))
    , nfaces(std::move(i_nfaces))
  {
  }

  const real* __restrict__ di_vertices;
  const int* __restrict__ di_faces;
  const real* __restrict__ di_face_area;
  const real* __restrict__ di_rho;
  const int32_t nverts;
  const int32_t nfaces;

  __host__ __device__ flo::real4
  operator()(thrust::tuple<int, const int> id) const
  {
    const int vid = id.get<0>();
    const int fid = id.get<1>();
    // Remove vid from faces[fid]
    int2 e_id;
    {
      const int fx = di_faces[fid + 0 * nfaces];
      const int fy = di_faces[fid + 1 * nfaces];
      const int fz = di_faces[fid + 2 * nfaces];
      if (fx == vid)
        e_id = make_int2(fy, fz);
      if (fy == vid)
        e_id = make_int2(fx, fz);
      if (fz == vid)
        e_id = make_int2(fx, fy);
    }

    const real ex =
      di_vertices[e_id.y + 0 * nverts] - di_vertices[e_id.x + 0 * nverts];
    const real ey =
      di_vertices[e_id.y + 1 * nverts] - di_vertices[e_id.x + 1 * nverts];
    const real ez =
      di_vertices[e_id.y + 2 * nverts] - di_vertices[e_id.x + 2 * nverts];

    const real rho = di_rho[vid];
    const real area = di_face_area[fid];

    flo::real4 o_val;
    o_val.x = o_val.y = o_val.z = 0.f;
    o_val.w = (ex * ex + ey * ey + ez * ez) / (4.f * area) +
              (rho * rho * area) * reciprocal(9.f);
    return o_val;
  }
};

template <
  typename T,
  typename = typename std::enable_if<std::is_same<T, real>::value ||
                                     std::is_same<T, real4>::value>::type>
void to_real_quaternion_matrix_impl(
  typename cusp::coo_matrix<int, T, cusp::device_memory>::const_view
    di_quaternion_matrix,
  cusp::array1d<int, cusp::device_memory>::const_view di_cumulative_column_size,
  cusp::coo_matrix<int, real, cusp::device_memory>::view do_real_matrix)
{
  dim3 block_dim;
  block_dim.z = 1;
  block_dim.y = 4;
  block_dim.x = 256;
  size_t nthreads_per_block = block_dim.x * block_dim.y * block_dim.z;
  size_t nblocks =
    di_quaternion_matrix.values.size() * 4 / nthreads_per_block + 1;
  size_t shared_memory_size =
    (sizeof(flo::real4) + sizeof(int32_t) * 2) * nthreads_per_block;

  d_to_real_quaternion_matrix<<<nblocks, block_dim, shared_memory_size>>>(
    di_quaternion_matrix.row_indices.begin().base().get(),
    di_quaternion_matrix.column_indices.begin().base().get(),
    di_quaternion_matrix.values.begin().base().get(),
    di_cumulative_column_size.begin().base().get(),
    di_quaternion_matrix.values.size(),
    do_real_matrix.row_indices.begin().base().get(),
    do_real_matrix.column_indices.begin().base().get(),
    do_real_matrix.values.begin().base().get());
  hipDeviceSynchronize();
}

}  // namespace

FLO_API void to_quaternion_matrix(
  cusp::coo_matrix<int, real4, cusp::device_memory>::const_view
    di_quaternion_matrix,
  cusp::array1d<int, cusp::device_memory>::const_view di_cumulative_column_size,
  cusp::coo_matrix<int, real, cusp::device_memory>::view do_real_matrix)
{
  to_real_quaternion_matrix_impl<real4>(
    di_quaternion_matrix, di_cumulative_column_size, do_real_matrix);
}

FLO_API void to_real_quaternion_matrix(
  cusp::coo_matrix<int, real, cusp::device_memory>::const_view
    di_quaternion_matrix,
  cusp::array1d<int, cusp::device_memory>::const_view di_cumulative_column_size,
  cusp::coo_matrix<int, real, cusp::device_memory>::view do_real_matrix)
{
  to_real_quaternion_matrix_impl<real>(
    di_quaternion_matrix, di_cumulative_column_size, do_real_matrix);
}

FLO_API void intrinsic_dirac(
  cusp::array2d<real, cusp::device_memory>::const_view di_vertices,
  cusp::array2d<int, cusp::device_memory>::const_view di_faces,
  cusp::array1d<real, cusp::device_memory>::const_view di_face_area,
  cusp::array1d<real, cusp::device_memory>::const_view di_rho,
  cusp::array2d<int, cusp::device_memory>::const_view di_entry_offset,
  cusp::array1d<int, cusp::device_memory>::const_view di_adjacency_keys,
  cusp::array1d<int, cusp::device_memory>::const_view di_adjacency,
  cusp::array1d<int, cusp::device_memory>::const_view di_cumulative_valence,
  cusp::array1d<int, cusp::device_memory>::const_view
    di_vertex_triangle_adjacency_keys,
  cusp::array1d<int, cusp::device_memory>::const_view
    di_vertex_triangle_adjacency,
  cusp::array1d<int, cusp::device_memory>::view do_diagonals,
  cusp::coo_matrix<int, real4, cusp::device_memory>::view do_dirac_matrix)
{
  // Find the diagonal matrix entry indices
  find_diagonal_indices(
    di_cumulative_valence, di_adjacency_keys, di_adjacency, do_diagonals);

  const int ndiagonals = do_diagonals.size();
  const int nnon_diagonals = do_dirac_matrix.num_entries - ndiagonals;

  // This will be used to permute the value iterator
  thrust::device_ptr<int> diagonal_stride_ptr{
    reinterpret_cast<int*>(do_dirac_matrix.values.begin().base().get())};
  auto diagonal_stride = cusp::make_array1d_view(
    diagonal_stride_ptr, diagonal_stride_ptr + nnon_diagonals);

  make_skip_indices(do_diagonals, diagonal_stride);
  // An iterator for each row, column pair of indices
  auto entry_it = thrust::make_zip_iterator(
    thrust::make_tuple(do_dirac_matrix.row_indices.begin(),
                       do_dirac_matrix.column_indices.begin()));
  // Iterator for non-diagonal matrix entries
  auto non_diag_begin =
    thrust::make_permutation_iterator(entry_it, diagonal_stride.begin());
  // Copy the adjacency keys and the adjacency info as the matrix coords
  thrust::copy_n(thrust::make_zip_iterator(thrust::make_tuple(
                   di_adjacency_keys.begin(), di_adjacency.begin())),
                 nnon_diagonals,
                 non_diag_begin);
  // Iterator for diagonal matrix entries
  auto diag_begin =
    thrust::make_permutation_iterator(entry_it, do_diagonals.begin());
  // Generate the diagonal entry, row and column indices
  thrust::tabulate(
    diag_begin, diag_begin + do_diagonals.size(), [] __device__(const int i) {
      return thrust::make_tuple(i, i);
    });

  const size_t block_width = 1024;
  const size_t nblocks = di_faces.num_cols * 4 / block_width + 1;
  // When passing the face and offset data to cuda, we reinterpret them as int
  // arrays. The advantage of this is coalesced memory reads by neighboring
  // threads, and access at a more granular level.
  // The cast is inherently safe due to the alignment of cuda vector types,
  // and reinterpret casting guarantees no changes to the underlying values
  d_intrinsic_dirac_atomic<<<nblocks, block_width>>>(
    di_vertices.values.begin().base().get(),
    di_faces.values.begin().base().get(),
    di_rho.begin().base().get(),
    di_entry_offset.values.begin().base().get(),
    di_vertices.num_cols,
    di_faces.num_cols,
    do_dirac_matrix.values.begin().base().get());
  hipDeviceSynchronize();

  // Iterate over adjacent faces and the corresponding vertex id
  auto face_vertex_iter = thrust::make_zip_iterator(
    thrust::make_tuple(di_vertex_triangle_adjacency_keys.begin(),
                       di_vertex_triangle_adjacency.begin()));

  // Transform opposing edge's, found through the vertex triangle adjacency
  // information, into diagonal dirac contributions
  // Doing this through the iterator saves a memory allocation
  auto dirac_iter = thrust::make_transform_iterator(
    face_vertex_iter,
    dirac_diagonal(di_vertices.values.begin().base(),
                   di_faces.values.begin().base(),
                   di_face_area.begin().base(),
                   di_rho.begin().base(),
                   di_vertices.num_cols,
                   di_faces.num_cols));

  thrust::reduce_by_key(
    di_vertex_triangle_adjacency_keys.begin(),
    di_vertex_triangle_adjacency_keys.end(),
    dirac_iter,
    thrust::make_discard_iterator(),
    thrust::make_permutation_iterator(do_dirac_matrix.values.begin(),
                                      do_diagonals.begin()));
}

FLO_DEVICE_NAMESPACE_END

