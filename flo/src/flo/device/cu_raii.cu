#include "flo/device/cu_raii.cuh"
#include <iostream>
#include <array>

FLO_DEVICE_NAMESPACE_BEGIN

namespace cu_raii
{
Stream::Stream()
{
  status = hipStreamCreate(&handle);
}

Stream::~Stream()
{
  join();
  status = hipStreamDestroy(handle);
}

Stream::operator hipStream_t() const noexcept
{
  return handle;
}

void Stream::join() noexcept
{
  status = hipStreamSynchronize(handle);
}

namespace solver
{
SolverSp::SolverSp()
{
  status = hipsolverSpCreate(&handle);
}

SolverSp::~SolverSp()
{
  hipsolverSpDestroy(handle);
}

SolverSp::operator hipsolverSpHandle_t() const noexcept
{
  return handle;
}

bool SolverSp::error_check(int line) const noexcept
{
  if (status == HIPSOLVER_STATUS_SUCCESS)
    return false;

  static constexpr std::array<const char*, 8> error_string = {
    "CUSOLVER_SUCCESS",
    "CUSOLVER_NOT_INITIALIZED",
    "CUSOLVER_ALLOC_FAILED",
    "CUSOLVER_INVALID_VALUE",
    "CUSOLVER_ARCH_MISMATCH",
    "CUSOLVER_EXECUTION_FAILED",
    "CUSOLVER_INTERNAL_ERROR",
    "CUSOLVER_MATRIX_TYPE_NOT_SUPPORTED"};

  std::cout << error_string[status];
  if (line != -1)
    std::cout << ", on line" << line;
  std::cout << '\n';
  return true;
}

void SolverSp::error_assert(int line) const noexcept
{
  if (error_check(line))
    std::exit(1);
}
}

namespace sparse
{
Handle::Handle()
{
  status = hipsparseCreate(&handle);
}

Handle::~Handle()
{
  hipsparseDestroy(handle);
}

Handle::operator hipsparseHandle_t() const noexcept
{
  return handle;
}

bool Handle::error_check(int line) const noexcept
{
  if (status == HIPSPARSE_STATUS_SUCCESS)
    return false;

  static constexpr std::array<const char*, 9> error_string = {
    "CUSPARSE_SUCCESS",
    "CUSPARSE_NOT_INITIALIZED",
    "CUSPARSE_ALLOC_FAILED",
    "CUSPARSE_INVALID_VALUE",
    "CUSPARSE_ARCH_MISMATCH",
    "CUSPARSE_MAPPING_ERROR",
    "CUSPARSE_EXECUTION_FAILED",
    "CUSPARSE_INTERNAL_ERROR",
    "CUSPARSE_MATRIX_TYPE_NOT_SUPPORTED"};

  std::cout << error_string[status];
  if (line != -1)
    std::cout << ", on line" << line;
  std::cout << '\n';
  return true;
}

void Handle::error_assert(int line) const noexcept
{
  if (error_check(line))
    std::exit(1);
}

MatrixDescription::MatrixDescription()
{
  hipsparseCreateMatDescr(&description);
}

MatrixDescription::MatrixDescription(hipsparseStatus_t* io_status)
{
  *io_status = hipsparseCreateMatDescr(&description);
}
MatrixDescription::~MatrixDescription()
{
  hipsparseDestroyMatDescr(description);
}

MatrixDescription::operator hipsparseMatDescr_t() const noexcept
{
  return description;
}
}  // namespace sparse

}  // namespace cu_raii

FLO_DEVICE_NAMESPACE_END
