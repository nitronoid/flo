#include "hip/hip_runtime.h"
#include "flo/device/similarity_xform.cuh"
#include <thrust/tabulate.h>
#include <cusp/monitor.h>
#include <cusp/krylov/cg.h>
#include <cusp/precond/diagonal.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform.h>
#include <thrust/scatter.h>
#include <cusp/permutation_matrix.h>

FLO_DEVICE_NAMESPACE_BEGIN

FLO_API void similarity_xform(
  cusp::coo_matrix<int, real, cusp::device_memory>::const_view di_dirac,
  cusp::array2d<real, cusp::device_memory>::view do_xform,
  const real i_tolerance,
  const int i_iterations)
{
  cu_raii::sparse::Handle sparse_handle;
  cu_raii::solver::SolverSp solver;

  similarity_xform(
    &sparse_handle, &solver, di_dirac, do_xform, i_tolerance, i_iterations);
}

FLO_API void similarity_xform(
  cu_raii::sparse::Handle* io_sparse_handle,
  cu_raii::solver::SolverSp* io_solver,
  cusp::coo_matrix<int, real, cusp::device_memory>::const_view di_dirac,
  cusp::array2d<real, cusp::device_memory>::view do_xform,
  const real i_tolerance,
  const int i_iterations)
{
  // TODO: FIX this and ammend the tests
  // Convert the row indices to csr row offsets
  cusp::array1d<int, cusp::device_memory> row_offsets(di_dirac.num_rows + 1);
  cusp::indices_to_offsets(di_dirac.row_indices, row_offsets);

  // Fill our initial guess with the identity (quaternions)
  thrust::tabulate(
    do_xform.values.begin(), do_xform.values.end(), [] __device__(int x) {
      // When x is a multiple of 4, return one
      return !(x & 3);
    });
  cusp::array1d<real, cusp::device_memory> b(do_xform.num_entries);

  // Get a cuSolver and cuSparse handle
  io_solver->error_assert(__LINE__);
  io_sparse_handle->error_assert(__LINE__);

  // Create a matrix description
  cu_raii::sparse::MatrixDescription description_D(&io_sparse_handle->status);
  io_sparse_handle->error_assert(__LINE__);

  // Tell cuSparse what matrix to expect
  hipsparseSetMatType(description_D, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatFillMode(description_D, HIPSPARSE_FILL_MODE_LOWER);
  hipsparseSetMatDiagType(description_D, HIPSPARSE_DIAG_TYPE_NON_UNIT);
  hipsparseSetMatIndexBase(description_D, HIPSPARSE_INDEX_BASE_ZERO);

  // Tell cusolver to use metis reordering
  const int reorder = 3;
  // cusolver will set this flag
  int singularity = 0;

  // Solve the system Dx = bx, using back substitution
  for (int iter = 0; iter < i_iterations + 1; ++iter)
  {
    // Get the norm of our right hand side
    const real rnorm = 1.f / cusp::blas::nrm2(do_xform.values);
    // Copy the result as our new right hand side, normalized
    thrust::transform(do_xform.values.begin(),
                      do_xform.values.end(),
                      b.begin(),
                      [=] __device__(real x) { return x * rnorm; });
    io_solver->status =
      hipsolverSpScsrlsvchol(*io_solver,
                            di_dirac.num_rows,
                            di_dirac.num_entries,
                            description_D,
                            di_dirac.values.begin().base().get(),
                            row_offsets.data().get(),
                            di_dirac.column_indices.begin().base().get(),
                            b.begin().base().get(),
                            i_tolerance,
                            reorder,
                            do_xform.values.begin().base().get(),
                            &singularity);
    io_solver->error_assert(__LINE__);
  }
  if (singularity != -1)
    std::cout << "Singularity: " << singularity << '\n';

  // Normalize the result and re-arrange simultaneously to reduce kernel the
  // number of launches
  {
    // Get the norm of our right hand side
    const real rnorm = 1.f / cusp::blas::nrm2(do_xform.values);
    // We re-arrange to place W as the last component of each vector
    auto scatter_out = thrust::make_permutation_iterator(
      do_xform.values.begin(),
      thrust::make_transform_iterator(
        thrust::make_counting_iterator(0),
        [w = do_xform.num_cols] __device__(int i) {
          // Transpose our index, and
          // simultaneously shuffle in the order:
          // x -> w
          // y -> x
          // z -> y
          // w -> z
          const int32_t x = (i + 3) & 3;
          const int32_t y = i >> 2;
          return x * w + y;
        }));
    thrust::transform(do_xform.values.begin(),
                      do_xform.values.end(),
                      scatter_out,
                      [=] __device__(real x) { return x * rnorm; });
  }
}

FLO_DEVICE_NAMESPACE_END
