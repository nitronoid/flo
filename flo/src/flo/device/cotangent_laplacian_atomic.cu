#include "hip/hip_runtime.h"
#include "flo/device/cotangent_laplacian_atomic.cuh"
#include "flo/device/thread_util.cuh"
#include <thrust/sort.h>
#include <thrust/reduce.h>

FLO_DEVICE_NAMESPACE_BEGIN

namespace
{
// block dim should be 3*#F, where #F is some number of faces,
// we have three edges per triangle face, and write two values per edge
__global__ void
d_cotangent_laplacian_atomic(const real3* __restrict__ di_vertices,
                             const int* __restrict__ di_faces,
                             const real* __restrict__ di_face_area,
                             const int* __restrict__ di_cumulative_valence,
                             const int* __restrict__ di_entry_offset,
                             const uint i_nfaces,
                             int* __restrict__ do_rows,
                             int* __restrict__ do_columns,
                             real* __restrict__ do_values)
{
  // Declare one shared memory block
  extern __shared__ uint8_t shared_memory[];
  // Create pointers into the block dividing it for the different uses
  real* __restrict__ cached_value = (real*)shared_memory;
  // There is a cached value for each corner of the face so we offset
  real3* __restrict__ points = (real3*)(cached_value + blockDim.x * 3);
  // There are nfaces *3 vertex values (duplicated for each face vertex)
  real* __restrict__ edge_norm2 = (real*)(points + blockDim.x * 3);
  // There are nfaces *3 squared edge lengths (duplicated for each face vertex)
  uint32_t* __restrict__ eid = (uint32_t*)(edge_norm2 + blockDim.x * 3);

  // Calculate which face this thread is acting on
  const uint fid = blockIdx.x * blockDim.x + threadIdx.x;

  // Check we're not out of range
  if (fid >= i_nfaces)
    return;

  // Get the vertex order, need to half the tid as we have two threads per edge
  const uchar3 loop = edge_loop(threadIdx.y >> 1);

  // Compute local edge indices rotated by the corner this thread corresponds to
  const uint16_t local_e0 = threadIdx.x * 3 + loop.x;
  const uint16_t local_e1 = threadIdx.x * 3 + loop.y;
  const uint16_t local_e2 = threadIdx.x * 3 + loop.z;

  // This thread will write to column or row major triangle based on even or odd
  const uint8_t major = !(threadIdx.y & 1);

  // Only write once per face
  if (!threadIdx.y)
  {
    // Duplicate for each corner of the face to reduce bank conflicts
    cached_value[local_e0] = cached_value[local_e1] = cached_value[local_e2] =
      di_face_area[fid] * 8.f;
  }
  // Write the vertex positions into shared memory
  if (major)
  {
    points[local_e0] = di_vertices[di_faces[fid * 3 + loop.x]];
  }
  __syncthreads();
  // Compute squared length of edges and write to shared memory
  if (major)
  {
    const real3 e = points[local_e2] - points[local_e1];
    edge_norm2[local_e0] = dot(e, e);
  }
  __syncthreads();
  if (major)
  {
    // Save the cotangent value into shared memory as multiple threads will,
    // write it into the final matrix
    cached_value[local_e0] =
      (edge_norm2[local_e1] + edge_norm2[local_e2] - edge_norm2[local_e0]) /
      cached_value[local_e0];
  }
  // Write the opposing edge ID's into shared memory to reduce global reads
  eid[local_e0 * 2 + !major] =
    di_faces[fid * 3 + nth_element(loop, 1 + !major)];
  __syncthreads();

  const uint32_t R = eid[local_e0 * 2 + !major];
  const uint32_t C = eid[local_e0 * 2 + major];
  const uint32_t address = di_entry_offset[fid * 6 + threadIdx.y] + R + (C > R);
  // Write the row and column indices
  do_rows[address] = R;
  do_columns[address] = C;
  atomicAdd(do_values + address, -cached_value[local_e0]);
}

}  // namespace

void cotangent_laplacian(
  const thrust::device_ptr<const real3> di_vertices,
  const thrust::device_ptr<const int3> di_faces,
  const thrust::device_ptr<const real> di_face_area,
  const thrust::device_ptr<const int> di_cumulative_valence,
  const thrust::device_ptr<const int2> di_entry_offset,
  const int i_nverts,
  const int i_nfaces,
  const int i_total_valence,
  thrust::device_ptr<int> do_diagonals,
  thrust::device_ptr<int> do_rows,
  thrust::device_ptr<int> do_columns,
  thrust::device_ptr<real> do_values)
{
  dim3 block_dim;
  block_dim.y = 6;
  block_dim.x = 170;
  size_t nthreads_per_block = block_dim.x * block_dim.y * block_dim.z;
  size_t nblocks = i_nfaces * 6 / nthreads_per_block + 1;
  // face area | cot_alpha  =>  sizeof(real) * 3 * #F
  // vertex positions       =>  sizeof(real3) * 3 * #F ==  sizeof(real) * 9 * #F
  // edge squared lengths   =>  sizeof(real) * 3 * #F
  // === (3 + 9 + 3) * #F * sizeof(real)
  size_t shared_memory_size =
    sizeof(flo::real) * block_dim.x * 15 + sizeof(uint32_t) * 6 * block_dim.x;

  // When passing the face and offset data to cuda, we reinterpret them as int
  // arrays. The advantage of this is coalesced memory reads by neighboring
  // threads, and access at a more granular level.
  // The cast is inherently safe due to the alignment of cuda vector types,
  // and reinterpret casting guarantees no changes to the underlying values
  d_cotangent_laplacian_atomic<<<nblocks, block_dim, shared_memory_size>>>(
    di_vertices.get(),
    reinterpret_cast<const int*>(di_faces.get()),
    di_face_area.get(),
    di_cumulative_valence.get(),
    reinterpret_cast<const int*>(di_entry_offset.get()),
    i_nfaces,
    do_rows.get(),
    do_columns.get(),
    do_values.get());
  hipDeviceSynchronize();

  thrust::counting_iterator<int> counter(0);
  thrust::for_each(counter+di_cumulative_valence[1],
                   counter + i_total_valence + i_nverts,
                   [do_diagonals = do_diagonals.get(),
                    do_rows = do_rows.get()] __device__(const int i) {
                     const int d = do_rows[i - 1];
                     if (d > do_rows[i])
                     {
                       do_diagonals[d] = i;
                     }
                   });

  // Iterator for diagonal matrix entries
  auto diag_begin = thrust::make_permutation_iterator(
    thrust::make_zip_iterator(thrust::make_tuple(do_rows, do_columns)),
    do_diagonals);

  // Generate the diagonal entry, row and column indices
  thrust::transform(
    counter, counter + i_nverts, diag_begin, [] __device__(const int i) {
      return thrust::make_tuple(i, i);
    });

  thrust::reduce_by_key(
    do_rows,
    do_rows + i_total_valence + i_nverts,
    thrust::make_transform_iterator(do_values, thrust::negate<int>()),
    thrust::make_discard_iterator(),
    thrust::make_permutation_iterator(do_values, do_diagonals));
}

FLO_DEVICE_NAMESPACE_END

