#include "hip/hip_runtime.h"
#include "test_common.h"
#include "device_test_util.h"
#include <cusp/print.h>
#include "flo/device/cotangent_laplacian.cuh"

TEST(CotangentLaplacian, cube)
{
  // cube faces all have area (1*1)/2 = 0.5
  std::vector<flo::real> h_area(12, 0.5);
  thrust::device_vector<flo::real> d_area = h_area;

  auto cube = make_cube();
  auto raw_vert_ptr = (flo::real3*)(&cube.vertices[0][0]);
  auto raw_face_ptr = (int3*)(&cube.faces[0][0]);

  thrust::device_vector<int3> d_faces(cube.n_faces());
  thrust::copy(raw_face_ptr, raw_face_ptr + cube.n_faces(), d_faces.data());

  thrust::device_vector<flo::real3> d_verts(cube.n_vertices());
  thrust::copy(raw_vert_ptr, raw_vert_ptr + cube.n_vertices(), d_verts.data());

  auto d_L = flo::device::cotangent_laplacian(
      d_verts.data(),
      d_faces.data(),
      d_area.data(),
      cube.n_vertices(),
      cube.n_faces());

  std::vector<int> I(d_L.row_indices.size());
  std::vector<int> J(d_L.column_indices.size());
  std::vector<flo::real> V(d_L.values.size());

  thrust::copy(d_L.row_indices.begin(), d_L.row_indices.end(), I.begin());
  thrust::copy(d_L.column_indices.begin(), d_L.column_indices.end(), J.begin());
  thrust::copy(d_L.values.begin(), d_L.values.end(), V.begin());


  std::vector<int> expected_I {
    0, 0, 0, 0, 0, 0, 
    1, 1, 1, 1, 1, 
    2, 2, 2, 2, 2,
    3, 3, 3, 3, 3, 3,
    4, 4, 4, 4, 4, 4,
    5, 5, 5, 5, 5, 
    6, 6, 6, 6, 6,
    7, 7, 7, 7, 7, 7
  };

  std::vector<int> expected_J {
    0, 1, 2, 4, 6, 7,
    0, 1, 2, 3, 7,
    0, 1, 2, 3, 4,
    1, 2, 3, 4, 5, 7,
    0, 2, 3, 4, 5, 6,
    3, 4, 5, 6, 7,
    0, 4, 5, 6, 7,
    0, 1, 3, 5, 6, 7
  };

  std::vector<flo::real> expected_V {
    3, -1, -1, -0, -1, -0,
   -1,  3, -0, -1, -1,
   -1, -0,  3, -1, -1,
   -1, -1,  3, -0, -1, -0,
   -0, -1, -0,  3, -1, -1,
   -1, -1,  3, -0, -1,
   -1, -1, -0,  3, -1,
   -0, -1, -0, -1, -1, 3 
  };

  using namespace testing;
  EXPECT_THAT(I, Pointwise(Eq(), expected_I));
  EXPECT_THAT(J, Pointwise(Eq(), expected_J));
  EXPECT_THAT(V, Pointwise(FloatNear(FLOAT_SOFT_EPSILON), expected_V));

}



