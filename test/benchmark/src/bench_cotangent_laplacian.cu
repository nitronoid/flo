#include "hip/hip_runtime.h"
#include <benchmark/benchmark.h>
#include <numeric>
#include "test_common.h"
#include "flo/device/area.cuh"
#include "flo/device/cotangent_laplacian.cuh"
#include "flo/device/vertex_vertex_adjacency.cuh"
#include "flo/host/valence.hpp"

#define DEVICE_BM_CLK(BM_NAME, FILE_NAME)                                      \
  static void BM_NAME(benchmark::State& state)                                 \
  {                                                                            \
    auto surf = TestCache::get_mesh(FILE_NAME);                                \
    auto raw_vert_ptr = (flo::real3*)(&surf.vertices[0][0]);                   \
    auto raw_face_ptr = (int3*)(&surf.faces[0][0]);                            \
    thrust::device_vector<int3> d_faces(surf.n_faces());                       \
    thrust::copy(raw_face_ptr, raw_face_ptr + surf.n_faces(), d_faces.data()); \
    thrust::device_vector<flo::real3> d_verts(surf.n_vertices());              \
    thrust::copy(                                                              \
      raw_vert_ptr, raw_vert_ptr + surf.n_vertices(), d_verts.data());         \
    auto d_area =                                                              \
      flo::device::area(d_verts.data(), d_faces.data(), d_faces.size());       \
    thrust::device_vector<int> d_valence(surf.n_vertices());                   \
    thrust::device_vector<int> d_cumulative_valence(surf.n_vertices() + 1);    \
    auto d_adjacency =                                                         \
      flo::device::vertex_vertex_adjacency(d_faces.data(),                     \
                                           surf.n_faces(),                     \
                                           surf.n_vertices(),                  \
                                           d_valence.data(),                   \
                                           d_cumulative_valence.data());       \
                                                                               \
    auto d_offsets =                                                           \
      flo::device::adjacency_matrix_offset(d_faces.data(),                     \
                                           d_adjacency.data(),                 \
                                           d_cumulative_valence.data(),        \
                                           surf.n_faces());                    \
    using SparseMatrix =                                                       \
      cusp::coo_matrix<int, flo::real, cusp::device_memory>;                   \
    SparseMatrix d_L(                                                          \
      surf.n_vertices(), surf.n_vertices(), d_cumulative_valence.back());      \
    thrust::fill(d_L.values.begin(), d_L.values.end(), 0);                     \
    dim3 block_dim;                                                            \
    block_dim.x = 3;                                                           \
    block_dim.y = 341;                                                         \
    size_t nthreads_per_block = block_dim.x * block_dim.y * block_dim.z;       \
    size_t nblocks = surf.n_faces() * 3 / nthreads_per_block + 1;              \
    size_t shared_memory_size = sizeof(flo::real) * block_dim.y * 15;          \
    for (auto _ : state)                                                       \
    {                                                                          \
      flo::device::d_cotangent_laplacian_atomic<<<nblocks,                     \
                                                  block_dim,                   \
                                                  shared_memory_size>>>(       \
        d_verts.data(),                                                        \
        thrust::device_ptr<const int>{(const int*)d_faces.data().get()},       \
        d_area.data(),                                                         \
        d_cumulative_valence.data(),                                           \
        thrust::device_ptr<const int>{(const int*)d_offsets.data().get()},     \
        surf.n_faces(),                                                        \
        d_L.column_indices.data(),                                             \
        d_L.row_indices.data(),                                                \
        d_L.values.data());                                                    \
      hipDeviceSynchronize();                                                 \
    }                                                                          \
  }                                                                            \
  BENCHMARK(BM_NAME)

#define DEVICE_BM_CLT(BM_NAME, FILE_NAME)                                      \
  static void BM_NAME(benchmark::State& state)                                 \
  {                                                                            \
    auto surf = TestCache::get_mesh(FILE_NAME);                                \
    auto vv = flo::host::valence(surf.faces);                                  \
    auto tv = std::accumulate(vv.begin(), vv.end(), 0);                        \
    auto raw_vert_ptr = (flo::real3*)(&surf.vertices[0][0]);                   \
    auto raw_face_ptr = (int3*)(&surf.faces[0][0]);                            \
    thrust::device_vector<int3> d_faces(surf.n_faces());                       \
    thrust::copy(raw_face_ptr, raw_face_ptr + surf.n_faces(), d_faces.data()); \
    thrust::device_vector<flo::real3> d_verts(surf.n_vertices());              \
    thrust::copy(                                                              \
      raw_vert_ptr, raw_vert_ptr + surf.n_vertices(), d_verts.data());         \
    auto d_area =                                                              \
      flo::device::area(d_verts.data(), d_faces.data(), d_faces.size());       \
    for (auto _ : state)                                                       \
    {                                                                          \
      benchmark::DoNotOptimize(                                                \
        flo::device::cotangent_laplacian(d_verts.data(),                       \
                                         d_faces.data(),                       \
                                         d_area.data(),                        \
                                         surf.n_vertices(),                    \
                                         surf.n_faces(),                       \
                                         tv));                                 \
    }                                                                          \
  }                                                                            \
  BENCHMARK(BM_NAME)

#define DEVICE_BM_CLA(BM_NAME, FILE_NAME)                                      \
  static void BM_NAME(benchmark::State& state)                                 \
  {                                                                            \
    auto surf = TestCache::get_mesh(FILE_NAME);                                \
    auto raw_vert_ptr = (flo::real3*)(&surf.vertices[0][0]);                   \
    auto raw_face_ptr = (int3*)(&surf.faces[0][0]);                            \
    thrust::device_vector<int3> d_faces(surf.n_faces());                       \
    thrust::copy(raw_face_ptr, raw_face_ptr + surf.n_faces(), d_faces.data()); \
    thrust::device_vector<flo::real3> d_verts(surf.n_vertices());              \
    thrust::copy(                                                              \
      raw_vert_ptr, raw_vert_ptr + surf.n_vertices(), d_verts.data());         \
    auto d_area =                                                              \
      flo::device::area(d_verts.data(), d_faces.data(), d_faces.size());       \
    thrust::device_vector<int> d_valence(surf.n_vertices());                   \
    thrust::device_vector<int> d_cumulative_valence(surf.n_vertices() + 1);    \
    auto d_adjacency =                                                         \
      flo::device::vertex_vertex_adjacency(d_faces.data(),                     \
                                           surf.n_faces(),                     \
                                           surf.n_vertices(),                  \
                                           d_valence.data(),                   \
                                           d_cumulative_valence.data());       \
                                                                               \
    auto d_offsets =                                                           \
      flo::device::adjacency_matrix_offset(d_faces.data(),                     \
                                           d_adjacency.data(),                 \
                                           d_cumulative_valence.data(),        \
                                           surf.n_faces());                    \
    for (auto _ : state)                                                       \
    {                                                                          \
      benchmark::DoNotOptimize(                                                \
        flo::device::cotangent_laplacian(d_verts.data(),                       \
                                         d_faces.data(),                       \
                                         d_area.data(),                        \
                                         d_cumulative_valence.data(),          \
                                         d_offsets.data(),                     \
                                         surf.n_vertices(),                    \
                                         surf.n_faces(),                       \
                                         d_cumulative_valence.back()));        \
    }                                                                          \
  }                                                                            \
  BENCHMARK(BM_NAME)

DEVICE_BM_CLT(DEVICE_cotangent_laplacian_cube_1, "../models/cube.obj");
DEVICE_BM_CLT(DEVICE_cotangent_laplacian_spot, "../models/spot.obj");
DEVICE_BM_CLT(DEVICE_cotangent_laplacian_sphere_400,
              "../models/dense_sphere_400x400.obj");
DEVICE_BM_CLT(DEVICE_cotangent_laplacian_sphere_1000,
              "../models/dense_sphere_1000x1000.obj");
DEVICE_BM_CLK(DEVICE_cotangent_laplacian_kernel_cube_1, "../models/cube.obj");
DEVICE_BM_CLK(DEVICE_cotangent_laplacian_kernel_spot, "../models/spot.obj");
DEVICE_BM_CLK(DEVICE_cotangent_laplacian_kernel_sphere_400,
              "../models/dense_sphere_400x400.obj");
DEVICE_BM_CLK(DEVICE_cotangent_laplacian_kernel_sphere_1000,
              "../models/dense_sphere_1000x1000.obj");
DEVICE_BM_CLA(DEVICE_cotangent_laplacian_atomic_cube_1, "../models/cube.obj");
DEVICE_BM_CLA(DEVICE_cotangent_laplacian_atomic_spot, "../models/spot.obj");
DEVICE_BM_CLA(DEVICE_cotangent_laplacian_atomic_sphere_400,
              "../models/dense_sphere_400x400.obj");
DEVICE_BM_CLA(DEVICE_cotangent_laplacian_atomic_sphere_1000,
              "../models/dense_sphere_1000x1000.obj");
// DEVICE_BM_CL(DEVICE_face_area_sphere_1500,
// "../models/dense_sphere_1500x1500.obj");
// DEVICE_BM_CL(DEVICE_face_area_cube_1000,
// "../models/cube_1k.obj");
